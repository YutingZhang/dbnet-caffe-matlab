#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/unpooling_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxUnpoolForward(const int nthreads, const Dtype* bottom_data, const Dtype* bottom_mask,
    const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    bottom_data += (n * channels + c) * height * width;
    bottom_mask += (n * channels + c) * height * width;
    top_data += (n * channels + c) * unpooled_height * unpooled_width;
    const int pool_idx = bottom_mask[h * width + w];
    top_data[pool_idx] = bottom_data[h * width + w];
  }
}

  // TO DO: debug average unpooling
template <typename Dtype>
__global__ void AveUnpoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % unpooled_width;
    int h = (index / unpooled_width) % unpooled_height;
    int c = (index / unpooled_width / unpooled_height) % channels;
    int n = index / unpooled_width / unpooled_height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, width);
    Dtype value = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, unpooled_height + pad_h);
        int wend = min(wstart + kernel_w, unpooled_width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        value += bottom_data[ph * width + pw] / pool_size;
      }
    }
    top_data[index] = value;
  }
}

// TO DO: need to test stochastic unpooling
template <typename Dtype>
__global__ void StoUnpoolForward(const int nthreads, const Dtype* bottom_data, Dtype* rand_idx,
    const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = h * stride_h;
    int hend = min(hstart + kernel_h, unpooled_height);
    int wstart = w * stride_w;
    int wend = min(wstart + kernel_w, unpooled_width); 
    float prob = 1.0 / static_cast<float>((hend - hstart + 1) * (wend - wstart + 1));
    top_data += (n * channels + c) * unpooled_height * unpooled_width;
    Dtype cumsum = 0.;
    float thres = rand_idx[index];
    for (int uh = hstart; uh < hend; ++uh) {
      for (int uw = wstart; uw < wend; ++uw) {
        cumsum += prob;
        if (cumsum >= thres) {
          rand_idx[index] = uh * unpooled_width + uw;
          top_data[uh * unpooled_width + uw] = bottom_data[index];
          return;
        }
      }
    } 
  }
}

template <typename Dtype>
void UnpoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = bottom[0]->count();
  caffe_gpu_set(top[0]->count(), Dtype(0.), top_data);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_bottom_mask = bottom.size() > 1;
  const Dtype* bottom_mask = NULL;
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnpoolingParameter_UnpoolMethod_MAX:
    if (use_bottom_mask) {
      bottom_mask = bottom[1]->gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxUnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom_mask, bottom[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case UnpoolingParameter_UnpoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AveUnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        top[0]->count(), bottom_data, bottom[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case UnpoolingParameter_UnpoolMethod_STOCHASTIC:
    caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
 			  rand_idx_.mutable_gpu_data());
    // NOLINT_NEXT_LINE(whitespace/operators)
    StoUnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, rand_idx_.mutable_gpu_data(), bottom[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxUnpoolBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_mask, const int num, const int channels,
    const int height, const int width, const int unpooled_height,
    const int unpooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    top_diff += (n * channels + c) * unpooled_height * unpooled_width;
    bottom_mask += (n * channels + c) * height * width;
    bottom_diff += (n * channels + c) * height * width;
    int pool_idx = bottom_mask[h * width + w];
    bottom_diff[h * width + w] = top_diff[pool_idx];
  }
}

  // TO DO: debug average unpooling
template <typename Dtype>
__global__ void AveUnpoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int pw = index % width;
    int ph = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, unpooled_height + pad_h);
    int wend = min(wstart + kernel_w, unpooled_width + pad_w);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, unpooled_height);
    wend = min(wend, unpooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * unpooled_height * unpooled_width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        gradient += top_diff[h * unpooled_width + w];
      }
    }
    bottom_diff[index] = gradient / pool_size;
  }
}

// TO DO: need to test stochastic unpooling
template <typename Dtype>
__global__ void StoUnpoolBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* rand_idx, const int num, const int channels,
    const int height, const int width, const int unpooled_height,
    const int unpooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    top_diff += (n * channels + c) * unpooled_height * unpooled_width;
    rand_idx += (n * channels + c) * height * width;
    int pool_idx = rand_idx[h * width + w];
    bottom_diff[index] = top_diff[pool_idx];
  }
}


template <typename Dtype>
void UnpoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll take as the input the mask in bottom[1] if it's of size >1.
  const bool use_bottom_mask = bottom.size() > 1;
  const Dtype* bottom_mask = NULL;
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnpoolingParameter_UnpoolMethod_MAX:
    if (use_bottom_mask) {
      bottom_mask = bottom[1]->gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxUnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_mask, top[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_diff);
    break;
  case UnpoolingParameter_UnpoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AveUnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), top_diff, top[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_diff);
    break;
  case UnpoolingParameter_UnpoolMethod_STOCHASTIC:
    // NOLINT_NEXT_LINE(whitespace/operators)
    StoUnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, rand_idx_.gpu_data(), top[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_diff);
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UnpoolingLayer);


}  // namespace caffe
