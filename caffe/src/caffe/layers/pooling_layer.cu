#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data, int* mask, Dtype* top_mask, bool use_local_idx) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart0 = ph * stride_h - pad_h;
    int wstart0 = pw * stride_w - pad_w;
    const int hend = min(hstart0 + kernel_h, height);
    const int wend = min(wstart0 + kernel_w, width);
    int hstart = max(hstart0, 0);
    int wstart = max(wstart0, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_slice[maxidx];
        }
      }
    }
    top_data[index] = maxval;

    int stored_maxidx = use_local_idx ?
  		  ( (maxidx/width-hstart0)*kernel_w+(maxidx%width-wstart0) ):(maxidx);
    if (mask) {
      mask[index] = stored_maxidx;
    } else {
      top_mask[index] = static_cast<Dtype>( stored_maxidx );
    }

  }
}

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
__global__
void SumPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype sumval = 0;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        sumval += bottom_slice[h * width + w];
      }
    }
    top_data[index] = sumval;
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* const bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* const rand_idx, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int hstart = ph * stride_h;
    const int hend = min(hstart + kernel_h, height);
    const int wstart = pw * stride_w;
    const int wend = min(wstart + kernel_w, width);
    Dtype cumsum = 0.;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
      }
    }
    const float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_slice[h * width + w];
          return;
        }
      }
    }
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* const bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int hstart = ph * stride_h;
    const int hend = min(hstart + kernel_h, height);
    const int wstart = pw * stride_w;
    const int wend = min(wstart + kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
        cumvalues += bottom_slice[h * width + w] * bottom_slice[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}


template <typename Dtype>
__global__ void FixPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int fix_x, const int fix_y,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int h = hstart + fix_y;
    int w = wstart + fix_x;
    if (h<0 || h>=height || w<0 || w>=width) {
    	top_data[index] = Dtype(0);
    } else {
        const Dtype* const bottom_slice =
            bottom_data + (n * channels + c) * height * width;
    	top_data[index] = bottom_slice[h * width + w];
    }

  }
}

template <typename Dtype>
__global__ void SwitchPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const Dtype* bottom_switch,
    Dtype* const top_data, bool use_local_idx) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    if (use_local_idx) {
		const int pw = index % pooled_width;
		const int ph = (index / pooled_width) % pooled_height;
		int hstart = ph * stride_h - pad_h;
		int wstart = pw * stride_w - pad_w;
		const int local_index = bottom_switch[index];
		int h = hstart+local_index/kernel_w;
		int w = wstart+local_index%kernel_w;
		if (h<0 || h>=height || w<0 || w>=width) {
			top_data[index] = Dtype(0);
		} else {
			const Dtype* const bottom_slice =
				bottom_data + (n * channels + c) * height * width;
			top_data[index] = bottom_slice[h * width + w];
		}
    } else {
	    const Dtype* const bottom_slice =
            bottom_data + (n * channels + c) * height * width;
    	const int bottom_index = bottom_switch[index];
    	top_data[index] = bottom_slice[ bottom_index ];
    }

  }
}


template <typename Dtype>
__global__
void SoftSwitchPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const Dtype* bottom_switch, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart0 = ph * stride_h - pad_h;
    int wstart0 = pw * stride_w - pad_w;
    int hend = min(hstart0 + kernel_h, height + pad_h);
    int wend = min(wstart0 + kernel_w, width + pad_w);
    int hstart = max(hstart0, 0);
    int wstart = max(wstart0, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;

    const int pooled_geo_count = pooled_width*pooled_height;
    const Dtype* switch_slice = bottom_switch +  (n*channels+c)*kernel_w*kernel_h*pooled_geo_count;
    const int slided_index = ph*pooled_width+pw;

    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
    	  int local_index  = (h-hstart0)*kernel_w+(w-wstart0);
    	  int switch_index = local_index*pooled_geo_count + slided_index;
          aveval += bottom_slice[h * width + w] * switch_slice[switch_index];
      }
    }
    top_data[index] = aveval;
  }
}


template <typename Dtype>
void PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,
        mask, top_mask, mask_index_type_ == PoolingParameter_MaskIndexType_LOCAL );
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case PoolingParameter_PoolMethod_SUM:
    // NOLINT_NEXT_LINE(whitespace/operators)
    SumPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    if (this->phase_ == TRAIN) {
      // We need to create the random index as well.
      caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                            rand_idx_.mutable_gpu_data());
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count),
                                   CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_,
          rand_idx_.mutable_gpu_data(), top_data);
    } else {
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, top_data);
    }
    break;
  case PoolingParameter_PoolMethod_FIX:
    // NOLINT_NEXT_LINE(whitespace/operators)
    FixPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        fix_x_, fix_y_,
        top_data);
    break;
  case PoolingParameter_PoolMethod_SWITCH:
    // NOLINT_NEXT_LINE(whitespace/operators)
  {
	const Dtype* bottom_switch = bottom[1]->gpu_data();
    SwitchPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_switch, top_data, mask_index_type_ == PoolingParameter_MaskIndexType_LOCAL);
  }
    break;
  case PoolingParameter_PoolMethod_SOFT_SWITCH:
    // NOLINT_NEXT_LINE(whitespace/operators)
  {
	const Dtype* bottom_switch = bottom[1]->gpu_data();
    SoftSwitchPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_switch, top_data);
  }
    break;

  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* const top_diff,
    const int* const mask, const Dtype* const top_mask, const int num,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h,
    const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
    const int pad_w, Dtype* const bottom_diff, Dtype* const bweights, bool use_local_idx ) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart =
         (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    const int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    const int pwstart =
         (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    Dtype gradient = 0; Dtype bwgt = 0;
    const int offset = (n * channels + c) * pooled_height * pooled_width;
    const Dtype* const top_diff_slice = top_diff + offset;
    if (use_local_idx) {
		if (mask) {
		  const int* const mask_slice = mask + offset;
		  for (int ph = phstart; ph < phend; ++ph) {
	    	int hstart = ph * stride_h - pad_h;
			for (int pw = pwstart; pw < pwend; ++pw) {
	    	  int wstart = pw * stride_w - pad_w;
	    	  int local_index = (h-hstart)*kernel_w+(w-wstart);
			  if (mask_slice[ph * pooled_width + pw] == local_index) {
				gradient += top_diff_slice[ph * pooled_width + pw];
				bwgt += Dtype(1.);
			  }
			}
		  }
		} else {
		  const Dtype* const top_mask_slice = top_mask + offset;
		  for (int ph = phstart; ph < phend; ++ph) {
    		int hstart = ph * stride_h - pad_h;
			for (int pw = pwstart; pw < pwend; ++pw) {
    		  int wstart = pw * stride_w - pad_w;
    		  int local_index = (h-hstart)*kernel_w+(w-wstart);
			  if (top_mask_slice[ph * pooled_width + pw] == local_index) {
				gradient += top_diff_slice[ph * pooled_width + pw];
				bwgt += Dtype(1.);
			  }
			}
		  }
		}
    } else {
		if (mask) {
		  const int* const mask_slice = mask + offset;
		  for (int ph = phstart; ph < phend; ++ph) {
			for (int pw = pwstart; pw < pwend; ++pw) {
			  if (mask_slice[ph * pooled_width + pw] == h * width + w) {
				gradient += top_diff_slice[ph * pooled_width + pw];
				bwgt += Dtype(1.);
			  }
			}
		  }
		} else {
		  const Dtype* const top_mask_slice = top_mask + offset;
		  for (int ph = phstart; ph < phend; ++ph) {
			for (int pw = pwstart; pw < pwend; ++pw) {
			  if (top_mask_slice[ph * pooled_width + pw] == h * width + w) {
				gradient += top_diff_slice[ph * pooled_width + pw];
				bwgt += Dtype(1.);
			  }
			}
		  }
		}
    }
    bottom_diff[index] = gradient;
    bweights[index] = bwgt;
  }
}

template <typename Dtype>
__global__ void AvePoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* const bottom_diff, Dtype* const bweights) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0; Dtype bwgt = 0;
    const Dtype* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
        bwgt += Dtype(1.)/Dtype(pool_size);
      }
    }
    bottom_diff[index] = gradient;
    bweights[index] = bwgt;
  }
}

template <typename Dtype>
__global__
void SumPoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* const bottom_diff, Dtype* const bweights) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0; Dtype bwgt = 0;
    const Dtype* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        gradient += top_diff_slice[ph * pooled_width + pw];
        bwgt += Dtype(1.);
      }
    }
    bottom_diff[index] = gradient;
    bweights[index] = bwgt;
  }
}



template <typename Dtype>
__global__ void StoPoolBackward(const int nthreads,
    const Dtype* const rand_idx, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* const bottom_diff, Dtype* const bweights) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0; Dtype bwgt = 0;
    const Dtype* const rand_idx_slice =
        rand_idx + (n * channels + c) * pooled_height * pooled_width;
    const Dtype* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff_slice[ph * pooled_width + pw] *
            (index == static_cast<int>(rand_idx_slice[ph * pooled_width + pw]));
        bwgt += (index == static_cast<int>(rand_idx_slice[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
    bweights[index] = bwgt;
  }
}


template <typename Dtype>
__global__ void FixPoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int fix_x, const int fix_y,
    Dtype* const bottom_diff, Dtype* const bweights) {
  CUDA_KERNEL_LOOP(index, nthreads) {
	// index is the pooled index (it is safe to do it here)
	const int pw = index % pooled_width;
	const int ph = (index / pooled_width) % pooled_height;
	const int c = (index / pooled_width / pooled_height) % channels;
	const int n = index / pooled_width / pooled_height / channels;
	int hstart = ph * stride_h - pad_h;
	int wstart = pw * stride_w - pad_w;
	int h = hstart + fix_y;
	int w = wstart + fix_x;
	if (!(h<0 || h>=height || w<0 || w>=width)) {
    	const int depooled_index = (n * channels + c) * height * width + h * width + w;
		bottom_diff[depooled_index] = top_diff[index];
		bweights[depooled_index] = Dtype(1.);
	}
  }
}


template <typename Dtype>
__global__
void SoftSwitchPoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const Dtype* const bottom_switch, Dtype* const bottom_diff, Dtype* const bweights) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0; Dtype bwgt = 0;

    const int pooled_geo_count = pooled_height * pooled_width;
    const int ch_count = n * channels + c;
    const int kernel_count = kernel_w * kernel_h;

    const Dtype* const top_diff_slice =
        top_diff + ch_count * pooled_geo_count;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;

        int local_index = (h-hstart)*kernel_w+(w-wstart);
        int top_sliced_index = ph * pooled_width + pw;
        int switch_index =  (ch_count * kernel_count + local_index) * pooled_geo_count + top_sliced_index;

        gradient += top_diff_slice[top_sliced_index] * bottom_switch[switch_index];
        bwgt += bottom_switch[switch_index];
      }
    }
    bottom_diff[index] = gradient;
    bweights[index] = bwgt;
  }
}


template <typename Dtype>
void PoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  Dtype* bweights = backward_weights_.mutable_gpu_data();

  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_SWITCH:
	top_mask = bottom[1]->gpu_data();
  case PoolingParameter_PoolMethod_MAX:
	if (!top_mask) {
		if (use_top_mask) {
		  top_mask = top[1]->gpu_data();
		} else {
		  mask = max_idx_.gpu_data();
		}
	}
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, top_mask, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_diff, bweights, mask_index_type_ == PoolingParameter_MaskIndexType_LOCAL);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff, bweights);
    break;
  case PoolingParameter_PoolMethod_SUM:
    // NOLINT_NEXT_LINE(whitespace/operators)
    SumPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff, bweights);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    // NOLINT_NEXT_LINE(whitespace/operators)
    StoPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, rand_idx_.gpu_data(), top_diff,
        top[0]->num(), channels_, height_, width_, pooled_height_,
        pooled_width_, kernel_h_, kernel_w_, stride_h_, stride_w_,
        bottom_diff, bweights);
    break;
  case PoolingParameter_PoolMethod_FIX:
    // NOLINT_NEXT_LINE(whitespace/operators)
    {
	const int top_count = top[0]->count();
	caffe_gpu_set(count, Dtype(0.), bottom_diff);
	caffe_gpu_set(count, Dtype(0.), bweights);
    FixPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        top_count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        fix_x_, fix_y_,
        bottom_diff, bweights);
    }
    break;
  case PoolingParameter_PoolMethod_SOFT_SWITCH:
    // NOLINT_NEXT_LINE(whitespace/operators)
  {
	const Dtype* bottom_switch = bottom[1]->gpu_data();
    SoftSwitchPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_switch, bottom_diff, bweights);
  }
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(PoolingLayer);


}  // namespace caffe
